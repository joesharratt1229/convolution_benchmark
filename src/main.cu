#include "hip/hip_runtime.h"
#include <string>
#include <cmath>

#include "common.h" 
#include "convolution.cuh"

using namespace std;

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


template<typename T>
__host__ void randomizeFilters(T h_filters[Nn][Ni][Ky][Kx]);
template<typename T>
__host__ void randomizeInput(T h_input[Ni][NyPad][NxPad]);
template<typename T>
__host__ void padInput(T h_input[Ni][NyPad][NxPad]);
template<typename T>
__host__ void printParameters();
template<typename T>
__host__ void convolution_cpu(T h_input[Ni][NyPad][NxPad], T h_filters[Nn][Ni][Ky][Kx], T h_output_cpu[Nn][Oy][Ox]);
template<typename T>
__host__ void checkOutput(T *h_output, T *h_output_cpu, unsigned int total_size);


int main(int argc, char **argv) {
    bool DEBUG = ((argc > 1) && (std::string(argv[1]) == "--debug"));

    unsigned int Ox2 = (Ox + 1) / 2;

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE, CHANNEL_SIZE);
    dim3 blocksPerGrid((Ox2 + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (Oy + threadsPerBlock.y - 1) / threadsPerBlock.y,
                       (Nn + threadsPerBlock.z - 1) / threadsPerBlock.z);


    static floatT h_input[Ni][NyPad][NxPad];
    static floatT h_output[Nn][Oy][Ox];
    static floatT h_output_cpu[Nn][Oy][Ox];
    static floatT h_filters[Nn][Ni][Ky][Kx]; 

    floatT (*d_input)[NyPad][NxPad];
    floatT (*d_output)[Oy][Ox];
    floatT (*d_filters)[Ni][Ky][Kx];


    hipMalloc((void**)&d_input, I_MEM_SIZE);
    hipMalloc((void**)&d_output, O_MEM_SIZE);
    hipMalloc((void**)&d_filters, F_MEM_SIZE);


    // Randomize inputs/filters and set padded regions to 0
    randomizeFilters(h_filters);
    randomizeInput(h_input);
    padInput(h_input);

    // Copy filters and input : host -> device
    gpuErrchk(hipMemcpy(d_input, h_input, I_MEM_SIZE, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_filters, h_filters, F_MEM_SIZE, hipMemcpyHostToDevice));


    // Start timer and execute kernel
    hipStream_t stream;
    hipStreamCreate(&stream);


    conv_2d<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_filters, d_output);

    gpuErrchk(hipDeviceSynchronize());

    // Copy output : device -> host
    gpuErrchk(hipMemcpy(h_output, d_output, O_MEM_SIZE, hipMemcpyDeviceToHost));

    // Check output
    if (DEBUG) {
        convolution_cpu(h_input, h_filters, h_output_cpu);
        checkOutput(&h_output[0][0][0], &h_output_cpu[0][0][0], Ox * Oy * Nn);
    } 

    return 0;
} 


template<typename T>
__host__
void randomizeFilters(T h_filters[Nn][Ni][Ky][Kx]) {
    for (int yy = 0; yy < Ky; ++yy)
        for (int xx = 0; xx < Kx; ++xx)
            for (int nn = 0; nn < Nn; ++nn)
                for (int ni = 0; ni < Ni; ++ni)
                    h_filters[nn][ni][yy][xx] = static_cast<T>(static_cast<float>(rand()) / static_cast<float>(RAND_MAX) - 0.5f);
}

template<typename T>
__host__
void randomizeInput(T h_input[Ni][NyPad][NxPad]) {
    for (int ni = 0; ni < Ni; ++ni)
        for (int yy = 0; yy < NyPad; ++yy)
            for (int xx = 0; xx < NxPad; ++xx)
                h_input[ni][yy][xx] = static_cast<T>(static_cast<float>(rand()) / static_cast<float>(RAND_MAX) - 0.5f);
}

template<typename T>
__host__
void padInput(T h_input[Ni][NyPad][NxPad]) {
    // Set padded regions to 0
    for (int z = 0; z < Ni; z++) {
            for (int x = 0; x < NxPad; x++) {
                h_input[z][0][x] = 0;
                h_input[z][NyPad - 1][x] = 0;
            }
            for (int y = 0; y < NyPad; y++) {
                h_input[z][y][0] = 0;
                h_input[z][y][NxPad - 1] = 0;
            }
    }
}


template<typename T>
__host__
void printParameters() {
    printf("\n\n");
    printf("Padding: %d\n", Pad);
    printf("Stride (StrideX, StrideY): (%d, %d)\n", StrideX, StrideY);

    printf("\n\n");
    printf("Input dimensions (Nx, Ny, Ni): (%d, %d, %d)\n", Nx, Ny, Ni);
    printf("Input dimensions with Pad (Nx+%d, Ny+%d, Ni): (%d, %d, %d)\n", (2 * Pad), (2 * Pad), NxPad, NyPad,
           Ni);
    printf("Input number of elements: %dx%dx%d = %d\n", Nx, Ny, Ni, Nx * Ny * Ni);
    printf("Input memory size: %lu bytes\n", I_MEM_SIZE);

    printf("\n\n");
    printf("Output dimensions (Ox, Oy, Nn): (%d, %d, %d)\n", Ox, Oy, Nn);
    printf("Output number of elements: %dx%dx%d = %d\n", Ox, Oy, Nn, Ox * Oy * Nn);
    printf("Output memory size: %lu bytes\n", O_MEM_SIZE);

    printf("\n\n");
    printf("Weights dimensions (Kx, Ky, Ni, Nn): (%d, %d, %d, %d)\n", Kx, Ky, Ni, Nn);
    printf("Weights number of elements: %dx%dx%dx%d = %d\n", Kx, Ky, Ni, Nn, Kx * Ky * Ni * Nn);
    printf("Weights memory size: %lu bytes\n", F_MEM_SIZE);
}



template<typename T>
__host__
void convolution_cpu(T h_input[Ni][NyPad][NxPad], T h_filters[Nn][Ni][Ky][Kx], T h_output_cpu[Nn][Oy][Ox]) {
    for (int nn = 0; nn < Nn; ++nn) {
        for (int oy = 0; oy < Oy; ++oy) {
            for (int ox = 0; ox < Ox; ++ox) {
                T sum = 0.0f;
                for (int ni = 0; ni < Ni; ++ni) {
                    for (int ky = 0; ky < Ky; ++ky) {
                        for (int kx = 0; kx < Kx; ++kx) {
                            int iy = oy * StrideY + ky;
                            int ix = ox * StrideX + kx;
                            sum += h_input[ni][iy][ix] * h_filters[nn][ni][ky][kx];
                        }
                    }
                }
                h_output_cpu[nn][oy][ox] = sum;
            }
        }
    }
}


template<typename T>
__host__
void checkOutput(T *h_output, T *h_output_cpu, unsigned int total_size) {
    for (int i = 0; i < total_size; i++) {
        float gpu_val = static_cast<float>(h_output[i]);
        float cpu_val = static_cast<float>(h_output_cpu[i]);
        if (std::abs(gpu_val - cpu_val) > 1e-3) {
            printf("Mismatch at h_output[%d]: %f (CPU) vs %f (GPU)\n", i, cpu_val, gpu_val);
            exit(1);
        }
    }
}